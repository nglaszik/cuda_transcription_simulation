#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <chrono>
#include <random>
#include <hiprand.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <array>
#include <sstream>
#include <map>
#include <fstream>
#include <iterator>

#include <sys/types.h>
#include <sys/stat.h>

#include <boost/filesystem.hpp>

using namespace std;
namespace fs = boost::filesystem;

// init some values
char path_output_dir[200]="path_output_dir";
char mode[10]="mode";
int max_count = 400;
double step = 1.0;
double h = 4.0; // bandwidth for kde
double max_time = 3600.0; // 1 hour
int batch_size = 1000000;
double lower_limit = -5.0; // lower limit for parameters
double upper_limit = 2.0; // upper limit for parameters
double k_deg = -1.0;
int num_cells = 0;
int num_cpgs = 0;

string concatenate(std::string const& name, float i)
{
	stringstream s;
	s << name << i;
	return s.str();
}

int dirExists(const char *path)
{
	struct stat info;

	if(stat( path, &info ) != 0)
		return 0;
	else if(info.st_mode & S_IFDIR)
		return 1;
	else
		return 0;
}

tuple <fs::path, fs::path, fs::path> run_path_checks(fs::path path_outdir, int max_count, float max_time, float step, float h, float lower_limit, float upper_limit, float k_deg, fs::path mode_dir){
	// check to see if output_dir exists
	if (!dirExists(path_outdir.c_str())){
		printf("%s directory does not exist, please create\n", path_outdir.c_str());
		exit(0);
	}
	else {
		printf("%s directory exists\n", path_outdir.c_str());
	}
	
	string rundir_string =  concatenate("ncell", max_count) + concatenate("_time", max_time) + concatenate("_step", step) + concatenate("_h", h) + concatenate("_lower", lower_limit) + concatenate("_upper", upper_limit) + concatenate("_deg", k_deg);
	fs::path rundir (rundir_string);
	fs::path path_mode_dir = path_outdir / mode_dir;
	
	if (!dirExists(path_mode_dir.c_str())){
		printf("%s does not exist\n", path_mode_dir.c_str());
		int stat = mkdir(path_mode_dir.c_str(), 0775);
		if (!stat){
			printf("%s directory created successfully\n", path_mode_dir.c_str());
		}
		else {
			printf("%s directory could not be created\n", path_mode_dir.c_str());
			exit(0);
		}
	} else {
		printf("%s directory exists\n", path_mode_dir.c_str());
	}
	
	fs::path path_run_dir = path_mode_dir / rundir;
	
	if (!dirExists(path_run_dir.c_str())){
		printf("%s does not exist\n", path_run_dir.c_str());
		int stat = mkdir(path_run_dir.c_str(), 0775);
		if (!stat){
			printf("%s directory created successfully\n", path_run_dir.c_str());
		}
		else {
			printf("%s directory could not be created\n", path_run_dir.c_str());
			exit(0);
		}
	} else {
		printf("%s directory exists\n", path_run_dir.c_str());
	}
	
	fs::path filename_kdes ("kdes.bin");
	fs::path filename_parameters ("parameters.csv");
	fs::path filename_counts ("counts.csv");
	
	fs::path path_kdes = path_run_dir / filename_kdes;
	fs::path path_parameters = path_run_dir / filename_parameters;
	fs::path path_counts = path_run_dir / filename_counts;
	
	return make_tuple(path_kdes, path_parameters, path_counts);
	
}

__device__
auto k_gpu(double val)
{
	const double p=1.0 / std::sqrt( 2.0 * M_PI);
	return p * std::exp(- 0.5 * (val*val));
}

auto k(double val)
{
	static const double p=1.0 / std::sqrt( 2.0 * M_PI);
	return p * std::exp(- 0.5 * (val*val));
}

__device__
auto generate_kde_gpu(double *distributions, int *mrna_counts, int max_count, double h, int batch_size, int i_param_combination, int num_cells)
{
	const double x_0 = 0.0;
	const int Nx = max_count;
	const double x_limit = (double)max_count;
	const double p = 1.0 / (h * max_count);
	const double hx = (x_limit - x_0)/(Nx - 1);
	
	for(int i_x = 0; i_x < Nx; ++i_x)
	{
		int i_dist = i_param_combination * max_count + i_x;
		double x = x_0 + i_x * hx;
		double sum = 0;
		for (int i_cell = 0; i_cell < num_cells; i_cell++) {
			int i_cell_param_combination = i_cell * batch_size + i_param_combination;
			//printf("filling distribution for cell %i, param combination %i, total index %i\n", i_cell, i_param_combination, i_cell_param_combination);
			//if (i_param_combination == 0) printf("%i,", mrna_counts[i_cell_param_combination]);
			sum += k_gpu((x - (double)mrna_counts[i_cell_param_combination]) / h);
		}
		distributions[i_dist] = p * sum;
	}
};

__global__
void generate_kde_gpu_parallel(double *distributions, int *mrna_counts, int max_count, double h, int num_genes, int num_cells)
{
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
		
	for (int i_gene = index; i_gene < num_genes; i_gene+=stride) {
		
		generate_kde_gpu(distributions, mrna_counts, max_count, h, num_genes, i_gene, num_cells);
		
	}
	
};

auto generate_kde(double *distributions, int *mrna_counts, int max_count, double h, int num_genes, int i_gene, int num_cells)
{
	const double x_0 = 0.0;
	const int Nx = max_count;
	const double x_limit = (double)max_count;
	const double p = 1.0 / (h * max_count);
	const double hx = (x_limit - x_0)/(Nx - 1);
	
	for(int i_x = 0; i_x < Nx; ++i_x)
	{
		int i_dist = i_gene * max_count + i_x;
		double x = x_0 + i_x * hx;
		double sum = 0;
		for (int i_cell = 0; i_cell < num_cells; i_cell++) {
			int i_cell_gene = i_cell * num_genes + i_gene;
			sum += k((x - (double)mrna_counts[i_cell_gene]) / h);
		}
		distributions[i_dist] = p * sum;
	}
};

auto kde(double x_limit, std::vector<double> values, double h)
{
	const double x_0 = 0.0;
	const int len = values.size();
	//const int Nx = 100;
	const int Nx = x_limit;
	const double p = 1.0 / (h * len);
	const double hx = (x_limit - x_0)/(Nx - 1);
	
	std::vector<double> output;
	
	for(int i_x = 0; i_x < Nx; ++i_x)
	{
		double x = x_0 + i_x * hx;
		double sum = 0;
		for(int i = 0; i < len; i++)
			sum += k((x - values[i]) / h);
		output.push_back(p * sum);
	}
	return output;
};

void convertCharrtoFarr(char *charr, float *farr) {
	int num_values = 1;
	int i;
	for(i=0; charr[i] != '\0'; i++){
		num_values += (charr[i] == ',');
	}
	// Traverse the charr
	int j = 0;
	char holder_arr[20]="";
	for (i = 0; charr[i] != '\0'; i++) {
		// if charr[i] is ',' then split
		if (charr[i] == ','){
			if (strcmp(holder_arr, "") != 0){
				farr[j] = atof(holder_arr);
				strcpy(holder_arr, "");
			}
			j++;
		}
		else {
			holder_arr[j] = charr[i];
		}
	}
}

float randomFloat(float min, float max) {
   return ((max - min) * ((float)rand() / RAND_MAX)) + min;
}

int parseCommand(int argc, char **argv) {
	for(int i=1;i<argc;) {
		//printf("argv[%u] = %s\n", i, argv[i]);
		if (strcmp(argv[i], "-o") == 0){
			strcpy(path_output_dir, argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-mode") == 0){
			strcpy(mode, argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-mt") == 0){
			max_time=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-s") == 0){
			step=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-ll") == 0){
			lower_limit=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-ul") == 0){
			upper_limit=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-bs") == 0){
			batch_size=atoi(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-ncell") == 0){
			num_cells=atoi(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-ncpg") == 0){
			num_cpgs=atoi(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-h") == 0){
			h=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-d") == 0){
			k_deg=atof(argv[i+1]);
			i=i+2;
		}
		else if (strcmp(argv[i], "-mc") == 0){
			max_count=atoi(argv[i+1]);
			i=i+2;
		}
		else{
			return 0;
		}
	}
	return 1;

}

__device__
double generate(hiprandState* globalState, int ind)
{
	hiprandState localState = globalState[ind];
	double RANDOM = hiprand_uniform_double( &localState );
	globalState[ind] = localState;
	return RANDOM;
}

__device__
int determine_event_alt(double prob_event, double *probs, int len_probs)
{
	double sum_probs = 0.0;
	for(int i=0; i < len_probs; i++){
		sum_probs += probs[i];
	}
	
	double sum_norm = 0.0;
	for(int i=0; i < len_probs; i++){
		probs[i] = probs[i] / sum_probs;
		sum_norm += probs[i];
	}
	
	//printf("%f\n", sum_norm);
	
	double rand_sum = 0.0;
	int i = 0;
	while (rand_sum < prob_event){
		rand_sum += probs[i];
		i += 1;
	}
	
	// do some checks to make sure this is working
	if (probs[0] == 0.0 && i-1 == 0){
		printf("err:0\n");
	}
	
	if (probs[1] == 0.0 && i-1 == 1){
		printf("err:1\n");
	}
	
	if (probs[2] == 0.0 && i-1 == 2){
		printf("err:2\n");
	}
	
	if (probs[3] == 0.0 && i-1 == 3){
		printf("err:3\n");
	}
	
	return i - 1;
}

__device__
double determine_event(double dt_switch, double dt_express, double dt_methylate, double dt_degrade, int *i_event)
{
	if (dt_switch <= dt_express && dt_switch <= dt_methylate && dt_switch <= dt_methylate){
		// switch
		*i_event = 0;
		return dt_switch;
	}
	else if (dt_switch <= dt_express){
		// switch on-off
		*i_event = 1;
		return dt_switch;
	} else {
		// transcribe
		*i_event = 2;
		return dt_express;
	}
}

__global__
void setup_kernel(hiprandState * state, unsigned long seed, int N)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < N) hiprand_init ( seed, id, 0, &state[id] );
}

__global__
void simulate(double max_time, int num_cells, int num_cpgs, int param_to_effect, int i_batch, int batch_size, int num_combinations_current_batch, const int num_params, int max_count, double h, double *param_combinations, int *transcriptional_states, int *mrna_count, int *num_meth_cpgs, double *simulated_distributions, hiprandState* globalState){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
		
	for (int i_param_combination = index; i_param_combination < num_combinations_current_batch; i_param_combination+=stride) {
		
		if (i_param_combination % 100000 == 0){
			printf("processing batch combo %i...\n", i_param_combination);
		}
		
		// reset counts and states
		for (int i_cell = 0; i_cell < num_cells; i_cell++){
			int i_cell_param_combination = i_cell * batch_size + i_param_combination;
			mrna_count[i_cell_param_combination] = 0;
			transcriptional_states[i_cell_param_combination] = 0;
			num_meth_cpgs[i_cell_param_combination] = 0;
		}
		
		for (int i_count = 0; i_count < max_count; i_count++){
			int i_dist = i_param_combination * max_count + i_count;
			simulated_distributions[i_dist] = 0.0;
		}
		
		int cpgs_to_methylate = (int)((double)num_cpgs * param_combinations[i_param_combination * num_params + 7]);					// f_meth
		
		for (int i_cell = 0; i_cell < num_cells; i_cell++) {
			
			int i_cell_param_combination = i_cell * batch_size + i_param_combination;
			double time = 0.0;
			int iteration = 0;			
			// no longer using iterations... need to make sure we get to steady state
			
			while (time < max_time && mrna_count[i_cell_param_combination] < max_count) {
				
				double prob_switch;
				double prob_express;
				double prob_degrade;
				double prob_methylate;
				double cpg_effect;
				
				// degradation
				prob_degrade = (double)mrna_count[i_cell_param_combination] * param_combinations[i_param_combination * num_params + 3]; //degradation of mrna
				
				// methylation
				// 4 = effect_size
				// 5 = direction
				// 6 = k_meth
				// 7 = f_meth
				int cpgs_left_to_methylate = cpgs_to_methylate - num_meth_cpgs[i_cell_param_combination];														
				prob_methylate = (double)cpgs_left_to_methylate * param_combinations[i_param_combination * num_params + 6];					// k_meth
				double percent_methylated = (double)num_meth_cpgs[i_cell_param_combination] / (double)num_cpgs;
				
				if (param_combinations[i_param_combination * num_params + 5] == 1.0){														// direction
					// positive regulation
					// starts at 1, then increases as a percent of cpgs methylated
					// when f is reached, some max effect is reached, but unless it's 1 it'll never have full effect
					cpg_effect = 1.0 + (percent_methylated * param_combinations[i_param_combination * num_params + 4]);						// effect_size
				} else {
					// negative regulation
					// starts at 1, then decreases as a percent of cpgs methylated
					// when f is reached, some max effect is reached, but unless it's 1 it'll never have full effect
					cpg_effect = 1.0 - (percent_methylated * param_combinations[i_param_combination * num_params + 4]);						// effect_size
				}
				
				// transcription
				if (transcriptional_states[i_cell_param_combination] == 0){
					// gene is off
					if (param_to_effect == 0){
						prob_switch = param_combinations[i_param_combination * num_params + 0] * cpg_effect;								// k_on
					} else {
						prob_switch = param_combinations[i_param_combination * num_params + 0];												// k_on
					}
					prob_express = 0.0;
				} else {
					// gene is on
					if (param_to_effect == 1){
						prob_switch = param_combinations[i_param_combination * num_params + 1] * cpg_effect;								// k_off
						prob_express = param_combinations[i_param_combination * num_params + 2];											// k_tx
					}
					else if (param_to_effect == 3){
						prob_switch = param_combinations[i_param_combination * num_params + 1];												// k_off
						prob_express = param_combinations[i_param_combination * num_params + 2] * cpg_effect;								// k_tx
					} else {
						prob_switch = param_combinations[i_param_combination * num_params + 1];												// k_off
						prob_express = param_combinations[i_param_combination * num_params + 2];											// k_tx
					}
				}
				
				// determine which event occurs & timestep
				double dt = -log(generate(globalState, i_param_combination)) / (prob_switch + prob_express + prob_methylate + prob_degrade);
				double probs [4] = {prob_switch, prob_express, prob_methylate, prob_degrade};
				int len_probs = 4;
				double prob_event = generate(globalState, i_param_combination);
				int i_event = determine_event_alt(prob_event, probs, len_probs);
				
				time = time + dt;
				iteration++;
				
				if (time < max_time){
					if (i_event == 0){
						// switch
						if (transcriptional_states[i_cell_param_combination] == 1){
							transcriptional_states[i_cell_param_combination] = 0;
						} else {
							transcriptional_states[i_cell_param_combination] = 1;
						}
					} 
					else if (i_event == 1){
						// transcribe
						mrna_count[i_cell_param_combination]++;
					}
					else if (i_event == 2){
						// methylate
						num_meth_cpgs[i_cell_param_combination]++;
					}
					else {
						// degrade
						mrna_count[i_cell_param_combination]--;
					}
				}
			}
		}		
		generate_kde_gpu(simulated_distributions, mrna_count, max_count, h, batch_size, i_param_combination, num_cells);
		
	}
}

vector<vector<double>> cart_product (const vector<vector<double>>& v) {
	vector<vector<double>> s = {{}};
	for (const auto& u : v) {
		vector<vector<double>> r;
		for (const auto& x : s) {
			for (const auto y : u) {
				r.push_back(x);
				r.back().push_back(y);
			}
		}
		s = move(r);
	}
	return s;
}

// 3D to 1D and reverse
// x = i % width;
// y = (i / width)%height;
// z = i / (width*height);
//i = x + width*y + width*height*z;

// nvcc /home/data/nlaszik/cuda_simulation/code/cuda/create_distributions_methylation.cu -o /home/data/nlaszik/cuda_simulation/code/cuda/build/create_distributions_methylation -lcurand -lboost_filesystem -lboost_system -lineinfo

// /home/data/nlaszik/cuda_simulation/code/cuda/build/create_distributions_methylation -mt 10.0 -mc 400 -s 0.2 -h 2.0 -bs 1000000 -o /home/data/nlaszik/cuda_simulation/output/simulated_methylation -mode k_tx -ll -3.0 -ul 3.0 -d 0.0 -ncell 1000 -ncpg 10

int main(int argc, char** argv)
{
	
	if(!parseCommand(argc, argv)) {
		cout<<"Error in arguments..\n";
		exit(0);
	}
	
	printf("max count: %i\n", max_count);
	printf("number of cells: %i\n", num_cells);
	printf("number of cpgs: %i\n", num_cpgs);
	printf("max time in seconds: %f\n", max_time);
	printf("batch size: %i\n", batch_size);
	printf("step size: %f\n", step);
	printf("h: %f\n", h);
	
	if (strcmp(mode, "mode") == 0){
		printf("Please provide a mode. Options: no_np, no_knp, no_pnp, full_model.\n");
		exit(0);
	}
	
	// check directories
	fs::path path_outdir (path_output_dir);
	fs::path path_mode (mode);
	fs::path path_kdes;
	fs::path path_parameters;
	fs::path path_counts;
	tie(path_kdes, path_parameters, path_counts) = run_path_checks(path_outdir, max_count, max_time, step, h, lower_limit, upper_limit, k_deg, path_mode);
	
	// test 0.0
	double test = pow(10.0, -DBL_MAX);
	if (test == 0.0){
		printf("0.0 test success: %f\n", test);
	} else {
		printf("0.0 test failed: %f\n", test);
		exit(0);
	}
	
	// test inf
	double inf_test = 10.0 / 0.0;
	printf("inf test: %f\n", inf_test);
	
	// check gpu memory
	float free_m,total_m,used_m;
	size_t free_t,total_t;
	hipMemGetInfo(&free_t,&total_t);
	free_m =(float)free_t/1048576.0;
	total_m=(float)total_t/1048576.0;
	used_m=total_m-free_m;
	printf ("mem free %f MB, mem total %f MB, mem used %f MB\n", free_m, total_m, used_m);
	
	printf("number of cells: %i\n", num_cells);
	
	int *transcriptional_states, *mrna_count, *num_meth_cpgs;
	double *param_combinations, *simulated_distributions;
	
	// creating parameter combinations
	printf("creating parameter combinations...\n");
	const int num_params = 8;
	// these are rates / second 
	// max rate should be once every 5 seconds = 720.0/hour = 0.2/sec... for high range, maybe instead just do linear rate changes 0.195, 0.19, 0.185, ... etc
	// for low range, next is maybe 0.19, 0.18, 0.1
	
	// min rate should 5.0/hour = 0.005/sec... we can actually do smaller increments it seems maybe .0025?
	
	
	// perhaps we can do this on log scale?
	
	// scale between k_np and k_on/k_off should be quite different? 
	// also play around with making bimodal distributions with small k_on and large k_off
	// transformers for gene/gene interactions? a la protein-protein interaction?
	
	// since log range, we start with negatives
	// k_on, k_off, k_tx, k_deg, effect_size, direction, k_meth, f_meth
	double param_lower_limits[num_params] = {lower_limit, 		lower_limit, 	lower_limit, 	k_deg,	0.1,	0.0,	0.5,	0.2};
	double param_upper_limits[num_params] = {upper_limit, 		upper_limit, 	upper_limit, 	k_deg,	1.0,	1.0,	5.0,	1.0};
	int param_to_effect = 0;
	if (strcmp(mode, "k_on") == 0){
		param_to_effect = 0;
	}
	else if (strcmp(mode, "k_off") == 0){
		param_to_effect = 1;
	}
	else if (strcmp(mode, "k_tx") == 0){
		param_to_effect = 2;
	}
	else {
		printf("Invalid mode provided. Please designate as 'k_on', 'k_off', or 'k_tx'\n");
		exit(0);
	}
	
	double step_effect = 0.1;
	double step_meth = 0.5;
	double step_f = 0.2;
	
	double step_sizes[num_params] = {(double)(step), (double)(step), (double)(step), (double)(step), (double)(step_effect), (double)(step), (double)(step_meth), (double)(step_f)};
	
	vector<vector<double>> param_matrix(num_params);
	// create parameters combinations
	int num_param_combinations = 1;
	for (int i_param = 0; i_param < num_params; i_param++){
		int param_size = 0;
		if (i_param == 5){
			param_matrix[i_param].push_back(0.0);
			param_matrix[i_param].push_back(1.0);
			param_size = 2;
		}
		else {
			for (double param = param_lower_limits[i_param]; param <= param_upper_limits[i_param]; param += step_sizes[i_param]){
				if (i_param == 4  || i_param == 6 || i_param == 7){
					// methylation should not be log scale... we know the ranges of methylation so this works
					param_matrix[i_param].push_back(param);
				}
				else{
					param_matrix[i_param].push_back(pow(10.0, param));
				}
				param_size++;
			}
		}
		num_param_combinations *= param_size;
	}
	
	printf("number of param combinations: %i\n", num_param_combinations);
	
	vector<vector<double>> param_combinations_vector = cart_product(param_matrix);
	
	int num_batches;
	if (num_param_combinations <= batch_size) {
		batch_size = num_param_combinations;
		num_batches = 1;
	}
	else {
		num_batches = (int)ceil(num_param_combinations / batch_size) + 1;
	}
	
	printf("num batches: %i, final batch size: %i\n", num_batches, batch_size);
	
	hipMallocManaged(&transcriptional_states, batch_size * num_cells * sizeof(int));
	hipMallocManaged(&mrna_count, batch_size * num_cells * sizeof(int));
	hipMallocManaged(&num_meth_cpgs, batch_size * num_cells * sizeof(int));
	hipMallocManaged(&simulated_distributions, batch_size * max_count * sizeof(double));
	hipMallocManaged(&param_combinations, batch_size * num_params * sizeof(double));
	
	// set up to fit on gpu
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	unsigned seed = chrono::system_clock::now().time_since_epoch().count();
	default_random_engine generator (seed);
	
	// setup and allocate memory for hiprand
	int N = batch_size;
	int *y, *d_y;
	y = (int*)malloc(N * sizeof(int));
	
	hipMalloc(&d_y, N * sizeof(int));
	hipMemcpy(d_y, y, N * sizeof(int), hipMemcpyHostToDevice);
	
	hiprandState* devStates;
	hipMalloc (&devStates, N * sizeof(hiprandState));
	
	// Run kernel on the GPU
	int blockSize = 32;
	int numBlocks = (N + blockSize - 1) / blockSize;
	
	setup_kernel<<<numBlocks, blockSize>>>(devStates, seed, N);
	
	hipEventRecord(start);
	
	// open counts file
	FILE *outfile_kdes;
	outfile_kdes = fopen(path_kdes.c_str(), "wb");
	
	FILE *outfile_counts;
	outfile_counts = fopen(path_counts.c_str(), "w");
	
	// open params file
	FILE *outfile_parameters;
	outfile_parameters = fopen(path_parameters.c_str(), "w");//create a file
	fprintf(outfile_parameters, "on,off,tx,deg,eff,dir,meth,f_meth,\n");
	
	printf("successfully opened output files\n");
	
	for (int i_batch = 0; i_batch < num_batches; i_batch++){
		
		// assign params vector to gpu memory
		int i_param_combination = 0;
		for (int i_batch_combination = i_batch * batch_size; i_batch_combination < (i_batch + 1) * batch_size; i_batch_combination++){
			if (i_batch_combination < num_param_combinations) {
				for (int i_param = 0; i_param < num_params; i_param++){
					int i_param_combination_param = i_param_combination * num_params + i_param;
					param_combinations[i_param_combination_param] = param_combinations_vector[i_batch_combination][i_param];
					fprintf(outfile_parameters, "%f,", param_combinations[i_param_combination_param]);
				}
				fprintf(outfile_parameters, "\n");
				i_param_combination++;
			}
			else {
				break;
			}
		}
		
		printf("processing combination batch %i, num combinations: %i...\n", i_batch + 1, i_param_combination);
		simulate<<<numBlocks, blockSize>>>(max_time, num_cells, num_cpgs, param_to_effect, i_batch, batch_size, i_param_combination, num_params, max_count, h, param_combinations, transcriptional_states, mrna_count, num_meth_cpgs, simulated_distributions, devStates);
		
		hipEventRecord(stop);
		hipDeviceSynchronize();
		
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		
		printf("Elapsed seconds: %f\n", milliseconds/1000);
		
		double *pt;
		for (int i_batch_combination = 0; i_batch_combination < i_param_combination; i_batch_combination++){
			for (int i_count = 0; i_count < max_count; i_count++){
				int i_dist = i_batch_combination * max_count + i_count;
				pt = &simulated_distributions[i_dist];
				fwrite(pt, sizeof(double), 1, outfile_kdes);
			}
			for (int i_cell = 0; i_cell < num_cells; i_cell++){
				int i_cell_param_combination = i_cell * batch_size + i_batch_combination;
				fprintf(outfile_counts, "%i,", mrna_count[i_cell_param_combination]);
			}
			fprintf(outfile_counts, "\n");
		}
	}

	// Free memory
	param_combinations_vector.clear();
	hipFree(param_combinations);
	hipFree(simulated_distributions);
	hipFree(transcriptional_states);
	hipFree(mrna_count);
	hipFree(num_meth_cpgs);
	hipFree(devStates);
	hipFree(d_y);
	return 0;
}
